#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

#include "../include/gpu.h"

// input tensor shape (1, 5, FEATURE_MAP_SIZE)
// 5: 4(xywh) + 1(class)
// output shape (1, FEATURE_MAP_SIZE, 5)
__global__ void transform_results(float *input_buffer, float *output_buffer, uint16_t FEATURE_MAP_SIZE)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x < FEATURE_MAP_SIZE)
    {
        for (int i = 0; i < 5; i++)
        {
            output_buffer[x * 5 + i] = input_buffer[i * FEATURE_MAP_SIZE + x];
        }
    }
}

PostProcess::PostProcess() {}

PostProcess::PostProcess(uint16_t max_detect, float conf_threshold, float iou_threshold, uint16_t feature_map_size) : MAX_DETECT(max_detect), CONF_THRESHOLD(conf_threshold), IOU_THRESHOLD(iou_threshold), FEATURE_MAP_SIZE(feature_map_size) {}

uint16_t PostProcess::init()
{
    check_status(hipMalloc(&this->transformed, FEATURE_MAP_SIZE * 5 * sizeof(float)));
    check_status(hipMalloc(&this->indices, FEATURE_MAP_SIZE * sizeof(int)));

    this->d_transformed = thrust::device_ptr<float>(this->transformed);
    this->d_indices = thrust::device_ptr<int>(this->indices);

    check_status(hipHostMalloc(&this->host_transformed, FEATURE_MAP_SIZE * 5 * sizeof(float)));
    check_status(hipHostMalloc(&this->host_indices, FEATURE_MAP_SIZE * sizeof(int)));

    return (uint16_t)hipSuccess;
}

uint16_t PostProcess::uninit()
{
    check_status(hipFree(this->transformed));
    check_status(hipFree(this->indices));

    check_status(hipHostFree(this->host_transformed));
    check_status(hipHostFree(this->host_indices));

    return (uint16_t)hipSuccess;
}

bool PostProcess::check_iou(float *box1, float *box2)
{
    float x1 = box1[0];
    float y1 = box1[1];
    float w1 = box1[2];
    float h1 = box1[3];
    float x2 = box2[0];
    float y2 = box2[1];
    float w2 = box2[2];
    float h2 = box2[3];
    float area_inter = fmax(fmin(x1 + w1 / 2, x2 + w2 / 2) - fmax(x1 - w1 / 2, x2 - w2 / 2), 0.0f) * fmax(fmin(y1 + h1 / 2, y2 + h2 / 2) - fmax(y1 - h1 / 2, y2 - h2 / 2), 0.0f);
    float area_union = w1 * h1 + w2 * h2 - area_inter;
    return area_inter / area_union > IOU_THRESHOLD;
}

// uint16_t PostProcess::post_process(float *input_buffer, float *output_buffer, uint16_t *num_detections)
// {
//     auto start = std::chrono::high_resolution_clock::now();
//     dim3 threads_pre_block(48, 2);
//     dim3 blocks(175);
//     transform_results<<<blocks, threads_pre_block>>>(input_buffer, this->transformed);
//     check_status(hipDeviceSynchronize());
//     auto end = std::chrono::high_resolution_clock::now();
//     auto diff = end - start;
//     std::cout << "Time taken by 1" << ": " << diff.count() << " seconds" << std::endl;
//     start = std::chrono::high_resolution_clock::now();
//     thrust::sequence(this->d_indices, this->d_indices + FEATURE_MAP_SIZE);
//     end = std::chrono::high_resolution_clock::now();
//     diff = end - start;
//     std::cout << "Time taken by 2" << ": " << diff.count() << " seconds" << std::endl;
//     start = std::chrono::high_resolution_clock::now();
//     thrust::sort(this->d_indices, this->d_indices + FEATURE_MAP_SIZE, [d_transformed = this->d_transformed] __device__(int a, int b)
//                  { return d_transformed[a * 16 + 4] > d_transformed[b * 16 + 4]; });
//     end = std::chrono::high_resolution_clock::now();
//     diff = end - start;
//     std::cout << "Time taken by 3" << ": " << diff.count() << " seconds" << std::endl;
//     start = std::chrono::high_resolution_clock::now();
//     check_status(hipMemcpy(this->host_indices, this->indices, FEATURE_MAP_SIZE * sizeof(int), hipMemcpyDeviceToHost));
//     check_status(hipMemcpy(this->host_transformed, this->transformed, FEATURE_MAP_SIZE * 16 * sizeof(float), hipMemcpyDeviceToHost));
//     end = std::chrono::high_resolution_clock::now();
//     diff = end - start;
//     std::cout << "Time taken by 4" << ": " << diff.count() << " seconds" << std::endl;
//     *num_detections = (uint16_t)MAX_DETECT;
//     start = std::chrono::high_resolution_clock::now();
//     int last = FEATURE_MAP_SIZE;
//     for (int i = 0; i < FEATURE_MAP_SIZE; ++i)
//     {
//         if (this->host_transformed[i * 16 + 4] < CONF_THRESHOLD)
//         {
//             last = i;
//             break;
//         }
//     }
//     for (int i = 0, j = 0; i < MAX_DETECT && j != -1; ++i)
//     {
//         int idx = this->host_indices[j];
//         if (this->host_transformed[idx * 16 + 4] < CONF_THRESHOLD)
//         {
//             *num_detections = (uint16_t)i;
//             break;
//         }
//         for (int item = 0; item < 16; ++item)
//         {
//             output_buffer[i * 16 + item] = this->host_transformed[idx * 16 + item];
//         }
//         int next = -1;
//         float *box = this->host_transformed + idx * 16;
//         for (; j < last; ++j)
//         {
//             int idx1 = this->host_indices[j];
//             if (idx1 == -1)
//             {
//                 continue;
//             }
//             if (check_iou(box, this->host_transformed + idx1 * 16))
//             {
//                 this->host_indices[j] = -1;
//             }
//             else if (next == -1)
//             {
//                 next = j;
//             }
//         }
//         j = next;
//     }
//     end = std::chrono::high_resolution_clock::now();
//     diff = end - start;
//     std::cout << "Time taken by 5" << ": " << diff.count() << " seconds" << std::endl;
//     return (uint16_t)hipSuccess;
// }

// input buffer (1, 5, FEATURE_MAP_SIZE)  (DEVICE)
// output buffer (MAX_DETECTION, 5)  (HOST)
// 5: 4(xywh) + 1(score)
uint16_t PostProcess::post_process(float *input_buffer, float *output_buffer, uint16_t *num_detections)
{
    dim3 threads_per_block(48);
    dim3 blocks((FEATURE_MAP_SIZE + 47) / 48);
    // (1, 5, FEATURE_MAP_SIZE)
    transform_results<<<blocks, threads_per_block>>>(input_buffer, this->transformed, FEATURE_MAP_SIZE);
    // (1, FEATURE_MAP_SIZE, 5)

    check_status(hipDeviceSynchronize());
    thrust::sequence(this->d_indices, this->d_indices + FEATURE_MAP_SIZE);
    thrust::sort(this->d_indices, this->d_indices + FEATURE_MAP_SIZE, [d_transformed = this->d_transformed] __device__(int a, int b)
                 { return d_transformed[a * 5 + 4] > d_transformed[b * 5 + 4]; });

    check_status(hipMemcpy(this->host_indices, this->indices, FEATURE_MAP_SIZE * sizeof(int), hipMemcpyDeviceToHost));
    check_status(hipMemcpy(this->host_transformed, this->transformed, FEATURE_MAP_SIZE * 16 * sizeof(float), hipMemcpyDeviceToHost));

    int end = FEATURE_MAP_SIZE;
    for (int i = 0; i < FEATURE_MAP_SIZE; ++i)
    {
        int idx = this->host_indices[i];
        if (this->host_transformed[idx * 5 + 4] < CONF_THRESHOLD)
        {
            end = i;
            break;
        }
    }

    if (end == 0)
    {
        *num_detections = 0;
        return (uint16_t)hipSuccess;
    }

    int i = 0;
    for (int j = 0; i < MAX_DETECT && j != -1; ++i)
    {
        int idx = this->host_indices[j];
        for (int item = 0; item < 16; ++item)
        {
            output_buffer[i * 16 + item] = this->host_transformed[idx * 16 + item];
        }

        int next = -1;
        float *box = this->host_transformed + idx * 16;
        for (; j < end; ++j)
        {
            int idx1 = this->host_indices[j];
            if (idx1 == -1)
            {
                continue;
            }
            if (check_iou(box, this->host_transformed + idx1 * 16))
            {
                this->host_indices[j] = -1;
            }
            else if (next == -1)
            {
                next = j;
            }
        }
        j = next;
    }
    *num_detections = (uint16_t)i;
    return (uint16_t)hipSuccess;
}

PostProcess *POSTPROCESS;

uint16_t postprocess_init_default()
{
    POSTPROCESS = new PostProcess();
    check_status(POSTPROCESS->init());
    return (uint16_t)hipSuccess;
}

uint16_t postprocess_init(uint16_t max_detect, float conf_threshold, float iou_threshold, uint16_t feature_map_size)
{
    POSTPROCESS = new PostProcess(max_detect, conf_threshold, iou_threshold, feature_map_size);
    check_status(POSTPROCESS->init());
    return (uint16_t)hipSuccess;
}

// input buffer (1, 32, FEATURE_MAP_SIZE)
// output buffer (MAX_DETECTION, 16)
// 16: 4(xywh) + 1(score) + 1(cls) + 10(kpnt)
uint16_t postprocess(float *input_buffer, float *output_buffer, uint16_t *num_detections)
{
    check_status(POSTPROCESS->post_process(input_buffer, output_buffer, num_detections));
    return (uint16_t)hipSuccess;
}

uint16_t postprocess_destroy()
{
    check_status(POSTPROCESS->uninit());
    delete POSTPROCESS;
    return (uint16_t)hipSuccess;
}

// input tensor shape (1, 5, FEATURE_MAP_SIZE)
// 5: 4(xywh) + 1(class)
// output shape (1, FEATURE_MAP_SIZE, 5)
__global__ void transform_results2(float *input_buffer, float *output_buffer, uint16_t FEATURE_MAP_SIZE)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x < FEATURE_MAP_SIZE)
    {
        float max_cls = input_buffer[4 * FEATURE_MAP_SIZE + x];
        int cls = 0;
        for (int i = 5; i < 16; i++)
        {
            float tmp = input_buffer[i * FEATURE_MAP_SIZE + x];
            if (max_cls < tmp)
            {
                max_cls = tmp;
                cls = i - 4;
            }
        }
        output_buffer[x * 2] = max_cls;
        output_buffer[x * 2 + 1] = (float)cls;
    }
}

uint16_t postprocess_classify(float *input_buffer, uint16_t feature_map_size, uint16_t *cls)
{
    static float *TRANSFORMED = nullptr;
    static int *INDICES = nullptr;
    
    if (TRANSFORMED == nullptr)
    {
        check_status(hipMalloc(&TRANSFORMED, FEATURE_MAP_SIZE * 2 * sizeof(float)));
        check_status(hipMalloc(&INDICES, FEATURE_MAP_SIZE * sizeof(int)));
    }

    d_TRANSFORMED = thrust::device_ptr<float>(TRANSFORMED);
    d_INDICES = thrust::device_ptr<int>(INDICES);

    dim3 threads_per_block(48);
    dim3 blocks((feature_map_size + 47) / 48);

    transform_results2<<<blocks, threads_per_block>>>(input_buffer, TRANSFORMED, FEATURE_MAP_SIZE);

    check_status(hipDeviceSynchronize());

    thrust::sequence(this->d_INDICES, this->d_INDICES + FEATURE_MAP_SIZE);
    thrust::sort(this->d_INDICES, this->d_INDICES + FEATURE_MAP_SIZE, [d_TRANSFORMED] __device__(int a, int b)
                 { return d_TRANSFORMED[a * 2] > d_TRANSFORMED[b * 2]; });

    int idx = 0;
    check_status(hipMemcpy(&idx, INDICES, sizeof(int), hipMemcpyDeviceToHost));
    float cls_f = 0;
    check_status(hipMemcpy(cls_f, TRANSFORMED + 2 * sizeof(float) * idx + 1, sizeof(float), hipMemcpyDeviceToHost));
    *cls = (uint16_t)cls_f;
}
