#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cfloat>
#include <cstdint>
#include "../include/gpu.h"

hipStream_t CUDASTREAM = nullptr;

// __global__ void rgbToTensor(unsigned char *input, float *output, uint32_t width, uint32_t height)
// {
//     int x = blockIdx.x * blockDim.x + threadIdx.x;
//     int y = blockIdx.y * blockDim.y + threadIdx.y;
//     if (x < width && y < height + 160)
//     {
//         int idx_out = (y * width + x);
//         int size_out = width * (height + 160);
//         if (y < height + 80 && y >= 80)
//         {
//             int idx_in = 3 * ((y - 80) * width + x);
//             output[idx_out] = input[idx_in] / 255.0f;                    // R
//             output[idx_out + size_out] = input[idx_in + 1] / 255.0f;     // G
//             output[idx_out + 2 * size_out] = input[idx_in + 2] / 255.0f; // B
//         }
//         else
//         {
//             output[idx_out] = 0.5f;                // R
//             output[idx_out + size_out] = 0.5f;     // G
//             output[idx_out + 2 * size_out] = 0.5f; // B
//         }
//     }
// }

// __global__ void rgbToTensor(uint8_t *input, float *output)
// {
//     int x = blockIdx.x * blockDim.x + threadIdx.x;
//     int y = blockIdx.y * blockDim.y + threadIdx.y;
//     if (x < 640 && y < 640)
//     {
//         int idx_out = (y * 640 + x);
//         int size_out = 640 * 640;
//         if (y < 560 && y >= 80)
//         {
//             int idx_in = 3 * ((y - 80) * 640 + x);
//             output[idx_out] = input[idx_in] / 255.0f;                    // R
//             output[idx_out + size_out] = input[idx_in + 1] / 255.0f;     // G
//             output[idx_out + 2 * size_out] = input[idx_in + 2] / 255.0f; // B
//         }
//         else
//         {
//             output[idx_out] = 0.5f;                // R
//             output[idx_out + size_out] = 0.5f;     // G
//             output[idx_out + 2 * size_out] = 0.5f; // B
//         }
//     }
// }

__global__ void rgbToTensor(uint8_t *input, float *output)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < 640 && y < 480)
    {
        int idx_out = (y * 640 + x);
        int size_out = 640 * 480;
        int idx_in = 3 * idx_out;

        output[idx_out] = input[idx_in] / 255.0f;                    // R
        output[idx_out + size_out] = input[idx_in + 1] / 255.0f;     // G
        output[idx_out + 2 * size_out] = input[idx_in + 2] / 255.0f; // B
    }
}

// assume that input is (640, 480, 3)
// output is (3, 640, 480)
// only normalize now
uint16_t convert_rgb888_3dtensor(uint8_t *input_buffer, float *output_buffer, uint32_t width, uint32_t height)
{
    dim3 threads_per_block(16, 16);
    dim3 num_blocks(40, 30);
    // rgbToTensor<<<num_blocks, threads_per_block, 0, CUDASTREAM>>>(input_buffer, output_buffer);
    // hipStreamSynchronize(CUDASTREAM);
    rgbToTensor<<<num_blocks, threads_per_block>>>(input_buffer, output_buffer);
    check_status(hipDeviceSynchronize());
    return (uint16_t)hipSuccess;
}

uint16_t transfer_host_to_device(uint8_t *host_mem, uint8_t *device_mem, uint32_t size)
{
    check_status(hipMemcpy(device_mem, host_mem, size, hipMemcpyHostToDevice));
    return (uint16_t)hipSuccess;
}

uint16_t transfer_device_to_host(uint8_t *host_mem, uint8_t *device_mem, uint32_t size)
{
    check_status(hipMemcpy(host_mem, device_mem, size, hipMemcpyDeviceToHost));
    return (uint16_t)hipSuccess;
}

uint16_t cuda_malloc(uint32_t size, uint8_t **buffer)
{
    check_status(hipMalloc((void **)buffer, size));
    return (uint16_t)hipSuccess;
}

uint16_t cuda_malloc_host(uint32_t size, uint8_t **buffer)
{
    check_status(hipHostMalloc((void **)buffer, size));
    return (uint16_t)hipSuccess;
}

uint16_t cuda_malloc_managed(uint32_t size, uint8_t **buffer)
{
    check_status(hipMallocManaged((void **)buffer, size));
    return (uint16_t)hipSuccess;
}

uint16_t cuda_free(uint8_t *buffer)
{
    check_status(hipFree(buffer));
    return (uint16_t)hipSuccess;
}

uint16_t cuda_free_host(uint8_t *buffer)
{
    check_status(hipHostFree(buffer));
    return (uint16_t)hipSuccess;
}

uint16_t init_cuda()
{
    check_status(hipStreamCreate(&CUDASTREAM));
    return (uint16_t)hipSuccess;
}

uint16_t destroy_cuda()
{
    check_status(hipStreamDestroy(CUDASTREAM));
    return (uint16_t)hipSuccess;
}